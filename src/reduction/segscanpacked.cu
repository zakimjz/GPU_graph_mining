#include "hip/hip_runtime.h"
#define NUM_THREADS PACKED_NUM_THREADS
#define BLOCKS_PER_SM PACKED_BLOCKS_PER_SM
#define VALUES_PER_THREAD PACKED_VALUES_PER_THREAD

#define NUM_WARPS (NUM_THREADS / WARP_SIZE)
#define LOG_NUM_WARPS LOG_BASE_2(NUM_WARPS)
#define VALUES_PER_WARP (WARP_SIZE * VALUES_PER_THREAD)
#define NUM_VALUES (NUM_THREADS * VALUES_PER_THREAD)


////////////////////////////////////////////////////////////////////////////////
// UPSWEEP PASS. Find the sum of all values in the last segment in each block.
// When the first head flag in the block is encountered, write out the sum to 
// that point and return. We only need to reduce the last segment to feed sums
// up to the reduction pass.

extern "C" __global__ __launch_bounds__(NUM_THREADS, BLOCKS_PER_SM)
void SegScanUpsweepPacked(const uint* packedIn_global, uint* blockLast_global,
	uint* headFlagPos_global, const int2* rangePairs_global) {

	uint tid = threadIdx.x;
	uint block = blockIdx.x;
	int2 range = rangePairs_global[block];

	const int UpsweepValues = 4;
	const int NumValues = UpsweepValues * NUM_THREADS;

	// Start at the last tile (NUM_VALUES before the end iterator). Because
	// upsweep isn't executed for the last block, we don't have to worry about
	// the ending edge case.
	int current = range.y - NumValues;

	uint threadSum = 0;
	int segmentStart = -1;

	while(current >= range.x) {

		uint packed[UpsweepValues];
	
		#pragma unroll
		for(int i = 0; i < UpsweepValues; ++i) 
			packed[i] = packedIn_global[current + tid + i * NUM_THREADS];

		// Find the index of the latest value loaded with a head flag set.
		int lastHeadFlagPos = -1;

		#pragma unroll
		for(int i = 0; i < UpsweepValues; ++i) {
			uint flag = 0x80000000 & packed[i];
			if(flag) lastHeadFlagPos = i;
		}
		if(-1 != lastHeadFlagPos)
			lastHeadFlagPos = tid + lastHeadFlagPos * NUM_THREADS;

		segmentStart = Reduce<NUM_WARPS>(tid, lastHeadFlagPos, 1);

		// Make a second pass and sum all the values that appear at or after
		// segmentStart.

		// Add if tid + i * NUM_THREADS >= segmentStart.
		// Subtract tid from both sides to simplify expression.
		int cmp = segmentStart - tid;
		#pragma unroll
		for(int i = 0; i < UpsweepValues; ++i) {
			uint value = 0x7fffffff & packed[i];
			if(i * NUM_THREADS >= cmp)
				threadSum += value;
		}

		if(-1 != segmentStart) break;

		__syncthreads();

		current -= NumValues;
	}

	__syncthreads();

	uint total = (uint)Reduce<NUM_WARPS>(tid, (int)threadSum, 0);

	if(0 == tid) {
		blockLast_global[block] = total;
		headFlagPos_global[block] = -1 != segmentStart;
	}
}


////////////////////////////////////////////////////////////////////////////////
// DOWNSWEEP PASS.

extern "C" __global__ __launch_bounds__(NUM_THREADS, BLOCKS_PER_SM)
void SegScanDownsweepPacked(const uint* packedIn_global, uint* valuesOut_global,
	const uint* start_global, const int2* rangePairs_global, int count,
	int inclusive) {

	uint tid = threadIdx.x;
	uint lane = (WARP_SIZE - 1) & tid;
	uint warp = tid / WARP_SIZE;
	uint block = blockIdx.x;
	uint index = VALUES_PER_WARP * warp + lane;

	int2 range = rangePairs_global[block];

	const int Size = NUM_WARPS * VALUES_PER_THREAD * (WARP_SIZE + 1);
	__shared__ volatile uint shared[Size];
	__shared__ volatile uint blockOffset_shared;

	// Use a stride of 33 slots per warp per value to allow conflict-free
	// transposes from strided to thread order.
	volatile uint* warpShared = shared + 
		warp * VALUES_PER_THREAD * (WARP_SIZE + 1);
	volatile uint* threadShared = warpShared + lane;

	// Transpose values into thread order.
	uint offset = VALUES_PER_THREAD * lane;
	offset += offset / WARP_SIZE;


	if(!tid) blockOffset_shared = start_global[block];


	while(range.x < range.y) {
		// Load values into packed.
		uint x[VALUES_PER_THREAD];
		uint flags[VALUES_PER_THREAD];

		////////////////////////////////////////////////////////////////////////
		// Load and transpose values.

        #pragma unroll
        for(int i = 0; i < VALUES_PER_THREAD; ++i) {
          uint source = range.x + index + i * WARP_SIZE;
          if(source < count) {
            uint packed = packedIn_global[source];
            threadShared[i * (WARP_SIZE + 1)] = packed;
          }
        }

		// Transpose into thread order and separate values from head flags.
		#pragma unroll
		for(int i = 0; i < VALUES_PER_THREAD; ++i) {
			uint packed = warpShared[offset + i];
			x[i] = 0x7fffffff & packed;
			flags[i] = 0x80000000 & packed;
		}

		////////////////////////////////////////////////////////////////////////
		// Run downsweep function on values and head flags.

		SegScanDownsweep<NUM_WARPS, VALUES_PER_THREAD>(tid, lane, warp, x, 
			flags, warpShared, threadShared, inclusive, &blockOffset_shared);

		////////////////////////////////////////////////////////////////////////
		// Transpose and store scanned values.

		#pragma unroll
		for(int i = 0; i < VALUES_PER_THREAD; ++i)
			warpShared[offset + i] = x[i];

        #pragma unroll
        for(int i = 0; i < VALUES_PER_THREAD; ++i) {
          uint target = range.x + index + i * WARP_SIZE;
          if(target < count) {
            valuesOut_global[target] = threadShared[i * (WARP_SIZE + 1)];
          }
        }

		range.x += NUM_VALUES;
	}
}


#undef NUM_THREADS
#undef NUM_WARPS
#undef LOG_NUM_WARPS
#undef BLOCKS_PER_SM
#undef VALUES_PER_THREAD
#undef VALUES_PER_WARP
#undef NUM_VALUES

