#include "hip/hip_runtime.h"
#define NUM_THREADS SCAN_NUM_THREADS
#define VALUES_PER_THREAD SCAN_VALUES_PER_THREAD
#define BLOCKS_PER_SM SCAN_BLOCKS_PER_SM


#define NUM_WARPS (NUM_THREADS / WARP_SIZE)
#define LOG_NUM_WARPS LOG_BASE_2(NUM_WARPS)
#define VALUES_PER_WARP (WARP_SIZE * VALUES_PER_THREAD)
#define NUM_VALUES (NUM_THREADS * VALUES_PER_THREAD)


////////////////////////////////////////////////////////////////////////////////
// Multiscan utility function. Used in the first and third passes of the
// global scan function. Returns the inclusive scan of the arguments in .x and
// the sum of all arguments in .y.

// Each warp is passed a pointer to its own contiguous area of shared memory.
// There must be at least 48 slots of memory. They should also be aligned so
// that the difference between the start of consecutive warps differ by an 
// interval that is relatively prime to 32 (any odd number will do).



////////////////////////////////////////////////////////////////////////////////
// GlobalScanUpsweep adds up all the values in elements_global within the 
// range given by blockCount and writes to blockTotals_global[blockIdx.x].

extern "C" __launch_bounds__(NUM_THREADS, BLOCKS_PER_SM) __global__ 
void GlobalScanUpsweep(const uint* valuesIn_global, uint* blockTotals_global, 
	const int2* range_global) {

	uint block = blockIdx.x;
	uint tid = threadIdx.x;
	int2 range = range_global[block];

	// Loop through all elements in the interval, adding up values.
	// There is no need to synchronize until we perform the multiscan.
	uint sum = 0;
	for(uint index = range.x + tid; index < range.y; index += 2 * NUM_THREADS)
		sum += valuesIn_global[index] + valuesIn_global[index + NUM_THREADS];

	// A full multiscan is unnecessary here - we really only need the total.
	// But this is easy and won't slow us down since this kernel is already
	// bandwidth limited.
	uint total = Multiscan2<NUM_WARPS>(tid, sum).y;

	if(!tid)
		blockTotals_global[block] = total;
}


////////////////////////////////////////////////////////////////////////////////
// GlobalScanReduction performs an exclusive scan on the elements in 
// blockTotals_global and writes back in-place.

extern "C" __global__ __launch_bounds__(REDUCTION_NUM_THREADS, 1)
void GlobalScanReduction(uint* blockTotals_global, 
	uint numBlocks) {

	uint tid = threadIdx.x;
	uint x = 0; 
	if(tid < numBlocks) x = blockTotals_global[tid];

	// Subtract the value from the inclusive scan for the exclusive scan.
	uint2 scan = Multiscan2<REDUCTION_NUM_THREADS / WARP_SIZE>(tid, x);
	if(tid < numBlocks) blockTotals_global[tid] = scan.x - x;

	// Have the first thread in the block set the scan total.
	if(!tid) blockTotals_global[numBlocks] = scan.y;
}


////////////////////////////////////////////////////////////////////////////////
// GlobalScanDownsweep runs an exclusive scan on the same interval of data as in
// pass 1, and adds blockScan_global[blockIdx.x] to each of them, writing back
// out in-place.

extern "C" __launch_bounds__(NUM_THREADS, BLOCKS_PER_SM) __global__ 
void GlobalScanDownsweep(const uint* valuesIn_global, uint* valuesOut_global,
	const uint* blockScan_global, const int2* range_global, int count, 
	int inclusive) {

	uint block = blockIdx.x;
	uint tid = threadIdx.x;
	uint warp = tid / WARP_SIZE;
	uint lane = (WARP_SIZE - 1) & tid;
	uint index = VALUES_PER_WARP * warp + lane;

	uint blockScan = blockScan_global[block];
	int2 range = range_global[block];

	const int Size = NUM_WARPS * VALUES_PER_THREAD * (WARP_SIZE + 1);
	__shared__ volatile uint shared[Size];

	// Use a stride of 33 slots per warp per value to allow conflict-free
	// transposes from strided to thread order.
	volatile uint* warpShared = shared + 
		warp * VALUES_PER_THREAD * (WARP_SIZE + 1);
	volatile uint* threadShared = warpShared + lane;

	// Transpose values into thread order.
	uint offset = VALUES_PER_THREAD * lane;
	offset += offset / WARP_SIZE;

	while(range.x < range.y) {

        #pragma unroll
        for(int i = 0; i < VALUES_PER_THREAD; ++i) {
            uint source = range.x + index + i * WARP_SIZE;
            uint x = valuesIn_global[source];
            if(i * (WARP_SIZE + 1) < count) {
              threadShared[i * (WARP_SIZE + 1)] = x;
            }
        }

		// Transpose into thread order by reading from transposeValues.
		// Compute the exclusive or inclusive scan of the thread values and 
		// their sum.
		uint scan[VALUES_PER_THREAD];
		uint sum = 0;
	
		#pragma unroll
		for(int i = 0; i < VALUES_PER_THREAD; ++i) {
			uint x = warpShared[offset + i];
			scan[i] = sum;
			if(inclusive) scan[i] += x;
			sum += x;
		}


		// Multiscan for each thread's scan offset within the block. Subtract
		// sum to make it an exclusive scan.
		uint2 localScan = Multiscan2<NUM_WARPS>(tid, sum);
		uint scanOffset = localScan.x + blockScan - sum;

		// Add the scan offset to each exclusive scan and put the values back
		// into the shared memory they came out of.
		#pragma unroll
		for(int i = 0; i < VALUES_PER_THREAD; ++i) {
			uint x = scan[i] + scanOffset;
			warpShared[offset + i] = x;
		}

		// Store the scan back to global memory.
		#pragma unroll
		for(int i = 0; i < VALUES_PER_THREAD; ++i) {
			uint x = threadShared[i * (WARP_SIZE + 1)];
			uint target = range.x + index + i * WARP_SIZE;
            if(target < count) {
              valuesOut_global[target] = x;
            }
		}

		// Grab the last element of totals_shared, which was set in Multiscan.
		// This is the total for all the values encountered in this pass.
		blockScan += localScan.y;

		range.x += NUM_VALUES;
	}
}


#undef NUM_THREADS
#undef NUM_WARPS
#undef LOG_NUM_WARPS
#undef BLOCKS_PER_SM
#undef VALUES_PER_THREAD
#undef VALUES_PER_WARP
#undef NUM_VALUES
