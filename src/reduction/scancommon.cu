#include "hip/hip_runtime.h"

////////////////////////////////////////////////////////////////////////////////
// Multiscan function parameterized over the number of warps in the block. Uses
// shared memory passed in from caller.

template<int NumWarps>
DEVICE2 uint2 Multiscan(uint tid, uint x, volatile uint* warpShared) {

	const int LogNumWarps = LOG_BASE_2(NumWarps);
		
	uint warp = tid / WARP_SIZE;
	uint lane = (WARP_SIZE - 1) & tid;

	volatile uint* s = warpShared + lane + WARP_SIZE / 2;
	warpShared[lane] = 0;
	s[0] = x;

	// Run inclusive scan on each warp's data.
	uint sum = x;

	#pragma unroll
	for(int i = 0; i < LOG_WARP_SIZE; ++i) {
		uint offset = 1<< i;
		sum += s[-offset];
		if(i < LOG_WARP_SIZE - 1) s[0] = sum;
	}

	__shared__ volatile uint totals_shared[2 * NumWarps];
	if(WARP_SIZE - 1 == lane) {
		totals_shared[NumWarps + warp] = sum;
	}

	// Synchronize to make all the totals available to the reduction code.
	__syncthreads();
	if(tid < NumWarps) {
		// Grab the block total for the tid'th block. This is the last element
		// in the block's scanned sequence. This operation avoids bank 
		// conflicts.
		uint total = totals_shared[NumWarps + tid];
		totals_shared[tid] = 0;
		volatile uint* s = totals_shared + NumWarps + tid;

		uint totalsSum = total;

		#pragma unroll
		for(int i = 0; i < LogNumWarps; ++i) {
			int offset = 1<< i;
			totalsSum += s[-offset];
			s[0] = totalsSum;	
		}

		// Subtract total from totalsSum for an exclusive scan.
		totals_shared[tid] = totalsSum - total;
	}

	// Synchronize to make the block scan available to all warps.
	__syncthreads();

	// Add the block scan to the inclusive sum for the block.
	sum += totals_shared[warp];
	uint total = totals_shared[2 * NumWarps - 1];
	return make_uint2(sum, total);
}


////////////////////////////////////////////////////////////////////////////////
// Multiscan that allocates its own shared memory. More convenient, but shared
// memory is limited, so only use when this doesn't result in an occupancy 
// decrease.

template<int NumWarps>
DEVICE2 uint2 Multiscan2(uint tid, uint x) {
	uint warp = tid / WARP_SIZE;
	const int WarpStride = WARP_SIZE + WARP_SIZE / 2;
	const int SharedSize = NumWarps * WarpStride;
	__shared__ volatile uint shared[SharedSize];
	volatile uint* warpShared = shared + warp * WarpStride;
	return Multiscan<NumWarps>(tid, x, warpShared);
}


////////////////////////////////////////////////////////////////////////////////
// Reduction function for upsweep pass. This performs addition for code 0 and
// max for code 1.

template<int NumWarps>
DEVICE2 int Reduce(uint tid, int x, int code) {

	const int LogNumWarps = LOG_BASE_2(NumWarps);

	uint warp = tid / WARP_SIZE;
	uint lane = (WARP_SIZE - 1) & tid;

	int init = code ? -1 : 0;

	const int ScanStride = WARP_SIZE + WARP_SIZE / 2 + 1;
	const int ScanSize = NumWarps * ScanStride;
	__shared__ volatile int reduction_shared[ScanSize];
	__shared__ volatile int totals_shared[2 * WARP_SIZE];

	volatile int* s = reduction_shared + ScanStride * warp + lane +
		WARP_SIZE / 2;
	s[-(WARP_SIZE / 2)] = init;
	s[0] = x;

	// Run intra-warp max reduction.
	#pragma unroll
	for(int i = 0; i < LOG_WARP_SIZE; ++i) {
		uint offset = 1<< i;
		if(0 == code) x += s[-offset];
		else if(1 == code) x = max(x, s[-offset]);
		s[0] = x;
	}

	// Synchronize to make all the totals available to the reduction code.
	__syncthreads();
	if(tid < NumWarps) {
		// Grab the block total for the tid'th block. This is the last element
		// in the block's scanned sequence. This operation avoids bank 
		// conflicts.
		x = reduction_shared[ScanStride * tid + WARP_SIZE / 2 + WARP_SIZE - 1];

		volatile int* s = totals_shared + NumWarps / 2 + tid;
		s[-(NumWarps / 2)] = init;
		s[0] = x;

		#pragma unroll
		for(int i = 0; i < LogNumWarps; ++i) {
			int offset = 1<< i;
			if(0 == code) x += s[-offset];
			else if(1 == code) x = max(x, s[-offset]);
			if(i < LogNumWarps - 1) s[0] = x;
		}
		totals_shared[tid] = x;
	}

	// Synchronize to make the block scan available to all warps.
	__syncthreads();

	return totals_shared[NumWarps - 1];
}


