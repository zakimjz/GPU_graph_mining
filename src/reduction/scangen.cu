#define WARP_SIZE 32
#define LOG_WARP_SIZE 5

// Use a 33-slot stride for shared mem transpose.
#define WARP_STRIDE (WARP_SIZE + 1)

typedef unsigned int uint;

#define DEVICE extern "C" __forceinline__ __device__ 
#define DEVICE2 __forceinline__ __device__

#define ROUND_UP(x, y) (~(y - 1) & (x + y - 1))

#include <hip/device_functions.h>
#include <vector_functions.h>

// Macro for computing LOG of NUM_WARPS


#define LOG_BASE_2(x) \
	((1 == x) ? 0 : \
		((2 == x) ? 1 : \
			((4 == x) ? 2 : \
				((8 == x) ? 3 : \
					((16 == x) ? 4 : \
						((32 == x) ? 5 : 0) \
					) \
				) \
			) \
		) \
	)

DEVICE uint bfi(uint x, uint y, uint bit, uint numBits) {
	uint ret;
	asm("bfi.b32 %0, %1, %2, %3, %4;" : 
		"=r"(ret) : "r"(y), "r"(x), "r"(bit), "r"(numBits));
	return ret;
}

#include "kernelparams.h"
#include "scancommon.cu"

#include "segscancommon.cu"

#include "globalscan.cu"
	
#include "segscanpacked.cu"
#include "segscanflags.cu"
#include "segscankeys.cu"
#include "segscanreduction.cu"

